#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/RunKernel.h"

__global__ void simple_vbo_kernel(float3 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width + x] = make_float3(u, w, v);
}
__global__ void color_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	float xx = x / (float)width;
	float yy = y / (float)height;
	float u = 0.5f+ 0.5f*cosf(time + xx + 0);
	float v = 0.5f+ 0.5f*cosf(time + yy + 2);
	float w = 0.5f + 0.5f*cosf(time + xx + 4);
	
	// write output vertex
	pos[y*width + x] = make_float4(u, v, w,1.0);
}
__global__ void grid_kernel(float3 * pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	float u = pos[x + y * width].x;
	float v = pos[x + y * width].z;
	float freq = 4.0f;
	pos[x + y * width].x = u;   
	pos[x + y * width].z = v;
	pos[x + y * width].y = 0.5*(u*u-(v-time)*(v-time));

}
__global__ void gerstenerKernel(float3 * pos, unsigned int width, unsigned int height, float amplitude,float time)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
	float u = (float)x /float(width);
	float v = (float)y / float(height);  
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	u += 0.01f;
	v += 0.01f;
	float dix1 = -(u) / (sqrt((u*u) +(v*v) ));
	float diy1 = -(v) / sqrt((u*u)+(v*v));
	float dix2 = (u - 0.45f) / (sqrt((u - 0.45f)*(u - 0.45f) + (v - 0.45f)*(v - 0.45f)));
	float diy2 = (v-0.45f) / sqrt((u - 0.45f)*(u - 0.45f) + (v - 0.45f)*(v - 0.45f));
	float wi = 0.7;
	float q = 1.7;
	float posx = u+ q*amplitude*dix1*cos((wi*((dix1)) +time)* 180 / 3.141592)+ q * amplitude*dix2*cos((wi*((dix2)) + time) * 180 / 3.141592);
	float posz = v+ q*amplitude*diy1*cos((wi*((diy1)) + time)* 180 / 3.141592)+ q * amplitude*diy2*cos((wi*((diy2)) + time) * 180 / 3.141592);
	float posy = amplitude*sin((wi*((dix1*u) + (diy1*v)) + time)*180 / 3.141592)+ amplitude * sin((wi*((dix2*u) + (diy2*v)) + time) * 180 / 3.141592);
	pos[x + width * y] = make_float3(posx,posy,posz);

 }
__global__ void multiWaveGerstenerKernelWithNormals(float3 * pos, float3 * norms ,unsigned int width, unsigned int height, WaveProp * prop, int n, float time)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
	float u = (float)x / float(width);
	float v = (float)y / float(height);
	u = 2.0f*u - 1.0f;
	v = 2.0f*v - 1.0f;
	u += 0.01f;
	v += 0.01f;
	float posx = u;
	float posz = v;
	float posy = 0;
	float normx = 0;
	float normy = 0;
	float normz = 0;
	for (int i = 0; i < n; i++)
	{
		float dix = 0.0f, diy = 0.0f;
		if (prop[i].isCircular == 0xFF)
		{
			float x = prop[i].x;
			float y = prop[i].y;
			x = u - x;
			y = v - y;
			dix = (x) / (sqrt(x*x + y * y));
			diy = (y) / (sqrt(y*y + y * y));
		}
		else
		{
			dix = prop[i].dirx;
			diy = prop[i].diry;
		}
		float amplitude = prop[i].amplitude;
		float q = prop[i].q;
		float wi = prop[i].w;
		float phi = prop[i].phase;
		posx += q * amplitude*dix*cos((wi*((dix)) + time * phi) * 180.0 / 3.141592);
		posz += q * amplitude*diy*cos((wi*((diy)) + time * phi) * 180.0 / 3.141592);
		posy += amplitude * sin((wi*((dix*u) + (diy*v)) + time * phi) * 180.0 / 3.141592);
		normx += dix * wi * amplitude * sin((wi*(dix*u + diy * v) + time * phi) * 180.0 / 3.141592);
		normz += diy * wi * amplitude * sin(((wi*(dix*u + diy * v)) + time * phi) * 180.0 / 3.141592);
		normy += q * wi * amplitude * cos((wi*(dix*u + diy * v) + time * phi) * 180.0 / 3.141592);
	}
	//normx -= normx;
	//normz -= normz;
	//normy = 1 - normy;
	pos[x + width * y] = make_float3(posx, posy, posz);
	norms[x + width * y] = make_float3(normx, normy, normz);

}
__global__ void multiWaveGerstenerKernel(float3 * pos,unsigned int width, unsigned int height, WaveProp * prop,int n ,float time)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
	float u = (float)x / float(width);
	float v = (float)y / float(height);
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	u += 0.01f;
	v += 0.01f;
	float posx = u;
	float posz = v;
	float posy = 0;
	for (int i = 0; i < n; i++)
	{
		float dix=0.0f, diy=0.0f;
		if (prop[i].isCircular == 0xFF)
		{
			float x = prop[i].x;
			float y = prop[i].y;
			x = u - x;
			y = v - y;
		    dix = (x) / (sqrt(x*x + y*y));
			diy = (y) / (sqrt(y*y + y*y));
		}
		else
		{
			dix = prop[i].dirx;
			diy = prop[i].diry;
		}
		float amplitude = prop[i].amplitude;
		float q = prop[i].q;
		float wi = prop[i].w;
		float phi = prop[i].phase;
		posx += q * amplitude*dix*cos((wi*((dix)) + time*phi) * 180 / 3.141592);
		posz += q * amplitude*diy*cos((wi*((diy)) + time*phi) * 180 / 3.141592);
	    posy += amplitude * sin((wi*((dix*u) + (diy*v)) + time*phi) * 180 / 3.141592);
	}
	pos[x + width * y] = make_float3(posx, posy, posz);
}
void UpdateMesh(float3 *pos, unsigned int mesh_width,
	unsigned int mesh_height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}
void GerstnerTest(float3 * pos, unsigned int mesh_width, unsigned int mesh_height, float amplitude,float time)
{   
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	gerstenerKernel << < grid, block >> > (pos, mesh_width, mesh_height,amplitude, time);
}
void GerstnerNormalTest(float3 * pos, float3 * norms,WaveProp * prop, MeshProp mesh, int n, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(mesh.mesh_width / block.x, mesh.mesh_height / block.y, 1);
	WaveProp * dev_ptr;
	checkCudaErrors(hipMalloc(&dev_ptr, sizeof(WaveProp)*n));
	checkCudaErrors(hipMemcpy(dev_ptr, prop, sizeof(WaveProp)*n, hipMemcpyHostToDevice));
	multiWaveGerstenerKernelWithNormals << <grid, block >> > (pos, norms,mesh.mesh_width, mesh.mesh_height, dev_ptr, n, time);
	getLastCudaError("Cuda Kernel Launch failed");
	checkCudaErrors(hipFree(dev_ptr));
}
void GerstnerTest(float3 * pos, WaveProp * prop, MeshProp mesh,int n ,float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(mesh.mesh_width / block.x, mesh.mesh_height / block.y, 1);
	WaveProp * dev_ptr;
	checkCudaErrors(hipMalloc(&dev_ptr,sizeof(WaveProp)*n));
	checkCudaErrors(hipMemcpy(dev_ptr, prop, sizeof(WaveProp)*n, hipMemcpyHostToDevice));
	multiWaveGerstenerKernel<< < grid, block >> > (pos, mesh.mesh_width, mesh.mesh_height, dev_ptr,n, time);
	getLastCudaError("Cuda Kernel Launch failed");
	checkCudaErrors(hipFree(dev_ptr));
}
void UpdateColors(float4 * pos, unsigned int width, unsigned int height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	color_kernel << < grid, block >> > (pos, width, height, time);
}

void UpdateGrid(float3 * pos, unsigned int width, unsigned int height, float time)
{
	dim3 block(16, 16, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	grid_kernel << <grid, block >> > (pos, width, height, time);
}
